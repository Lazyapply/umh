#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <omp.h>


#define N 256


__global__ void add2Vectors(int *v1, int *v2, int *vR){
	//indice del hilo
	int indexT = blockIdx.x * blockDim.x + threadIdx.x;
	vR[indexT] = v1[indexT] + v2[indexT];
}

int main(){

	int 	i, j; //iteradores
	int		**h_m1, **h_m2, **h_mR; //host
	int		*d_v1, *d_v2, *d_vR; //device
	double	t1CPU, t2CPU, t1GPU, t2GPU;


	//reservamos memoria
	h_m1 = (int**)malloc(N*sizeof(int*));
	h_m2 = (int**)malloc(N*sizeof(int*));
	h_mR = (int**)malloc(N*sizeof(int*));
	
	for(i=0;i<N;i++){
		h_m1[i] = (int*)malloc(N*sizeof(int));
		h_m2[i] = (int*)malloc(N*sizeof(int));
		h_mR[i] = (int*)malloc(N*sizeof(int));

		//inicializacion de las matrices
		for(j=0;j<N;j++){
			h_m1[i][j] = rand()%9;
			h_m2[i][j] = rand()%9;
			h_mR[i][j] = 0;
			//printf(" %d ", h_m1[i][j]);
		}
		//printf("\n");
	}

	//resultado CPU (secuencial)
	t1CPU = omp_get_wtime();
	for(i=0;i<N;i++){
		for(j=0;j<N;j++){
			h_mR[i][j] = h_m1[i][j] + h_m2[i][j];
		}
	}
	t2CPU = omp_get_wtime();

	int idDevice = 0;
	int deviceCount;

	hipDeviceProp_t gtxProp;
	
	if (hipInit(0) != 0){ // Obligatorio para que funcione la API
		printf("ERROR De Inicializacion\n");
		exit(0);
	}

	hipGetDeviceCount(&deviceCount); //Devuelve el nº de GPU's CUDA
	if (deviceCount == 0){
		printf("ERROR Ningun dispositivo es compatible con CUDA\n");
		exit(0);
	}

	//recogemos la estrucutra de propiedades
	hipGetDeviceProperties(&gtxProp, idDevice);

	//escribimos las propiedades del dispositivo
	printf("\n\tDispositivo CUDA Instalado: \"%s\"\n\n",gtxProp.name);
	printf("\tCapacidades de Cómputo: Revision( %d.%d )\n",gtxProp.major, gtxProp.minor);

	printf("\tCantidad TOTAL de Memoria GLOBAL: %ld MBytes\n", gtxProp.totalGlobalMem/1048576);
	printf("\tCantidad de Memoria Compartida por bloque: %ld KBytes\n", gtxProp.sharedMemPerBlock/1024);
	printf("\tTamaño de WARP: %d Threads\n",gtxProp.warpSize);
	printf("\tTamaño máximo de Hilos por Bloque: %d\n",gtxProp.maxThreadsPerBlock);

	printf("\n\t\tDIMENSIONES\n\n");
	printf("\tDimensiones maximas de bloque:  ( %d x %d x %d )\n", 
		gtxProp.maxThreadsDim[0],
		gtxProp.maxThreadsDim[1],
		gtxProp.maxThreadsDim[2]);
	printf("\tDimensiones maximas de grid:   ( %d x %d x %d )\n",
	   gtxProp.maxGridSize[0],
	   gtxProp.maxGridSize[1],
	   gtxProp.maxGridSize[2]);
	printf("\tFrecuencia de Reloj: %.3f GHz\n", (float)gtxProp.clockRate/1000000);
	printf("\tMemoria CONSTANT disponible: %ld Bytes\n", gtxProp.totalConstMem);


	//reservamos CUDA
	hipMalloc((void**)&d_v1, N*N * sizeof(d_v1));
	hipMalloc((void**)&d_v2, N*N * sizeof(d_v2));
	hipMalloc((void**)&d_vR, N*N * sizeof(d_vR));
	hipMemset(d_v1, 0, N*N*sizeof(d_v1));
	hipMemset(d_v2, 0, N*N*sizeof(d_v2));
	hipMemset(d_vR, 0, N*N*sizeof(d_vR));


	//copiamos la matriz en forma de vector
	//-----------------------------
	//apartado A)
	//-----------------------------
	int iterador = 0;
	for(i=0;i<N;i++){
		//copiamos el vector que vamos a usar en este momento
		hipMemcpy(&d_v1[iterador], h_m1[i], N*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(&d_v2[iterador], h_m1[i], N*sizeof(int), hipMemcpyHostToDevice);
		iterador += N - 1;
	}

	//tiempos
	t1GPU = omp_get_wtime();
	add2Vectors<<<128,512>>>(d_v1, d_v2, d_vR);
	hipDeviceSynchronize();
	t2GPU = omp_get_wtime();

	//resultados
	printf("\n\t Resultadosn--------------------------");
	printf("\n\t El tiempo CPU es %fs", t2CPU - t1CPU);
	printf("\n\t El tiempo GPU(A) es %fs", t2GPU - t1GPU);

	//-----------------------------
	//apartado B)
	//-----------------------------
	
	int *mA, *mB, *mR;
	int *d_2A, *d_2B, *d_2R;
	int tam = N * N;

	mA = (int*)malloc(tam*sizeof(int));
	mB = (int*)malloc(tam*sizeof(int));
	mR = (int*)malloc(tam*sizeof(int));

	for(i=0;i<tam;i++){
		mA[i] = rand()%9;
		mB[i] = rand()%9;
		mR[i] = 0;
	}

	//reserva de memoria
	hipMalloc((void**)&d_2A, tam*sizeof(d_2A));
	hipMemset(d_2A, 0, tam*sizeof(d_2A));
	hipMalloc((void**)&d_2B, tam*sizeof(d_2B));
	hipMemset(d_2B, 0, tam*sizeof(d_2B));
	hipMalloc((void**)&d_2R, tam*sizeof(d_2R));
	hipMemset(d_2R, 0, tam*sizeof(d_2R));

	//copiamos
	hipMemcpy(d_2A, mA, tam*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_2B, mB, tam*sizeof(int), hipMemcpyHostToDevice);

	//llamamos al kernel
	t1GPU = omp_get_wtime();
	add2Vectors<<<1024,64>>>(d_2A, d_2B, d_2R);
	t2GPU = omp_get_wtime();

	//resultados 2
	printf("\n\t El tiempo CPU es %fs", t2CPU - t1CPU);
	printf("\n\t El tiempo GPU(B) es %fs\n", t2GPU - t1GPU);

	hipMemcpy(mR, d_2R, tam*sizeof(int), hipMemcpyDeviceToHost);
}