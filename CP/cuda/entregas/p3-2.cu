#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#define N 			512
#define BLOCK_SIZE 	512

__global__ void internProd(int *intProdParcial, int *d1, int *d2){

	__shared__ double accumR[BLOCK_SIZE];

	int pos 	= blockIdx.x * blockDim.x + threadIdx.x;
	int iAccum	= threadIdx.x;
	int vec 	= blockIdx.x;

	accumR[iAccum] = d1[pos] * d1[pos];

	for(int stride = BLOCK_SIZE/2; stride>0;stride>>=1){
		__syncthreads();
		accumR[iAccum] += accumR[stride + iAccum];
	}

	if(threadIdx.x == 0){
		intProdParcial[vec] = accumR[0];
	}
}


int main(int argc, char *argv[]){

	int i, j;
	int **m, *v, *vR;
	int *d_m, *d_v, *d_vR;
	int *res;
	double t1GPU, t2GPU;

	//reservas en HOST
	m 	= (int**)malloc(N*sizeof(int*));
	v 	= (int*)malloc(N*sizeof(int));
	vR 	= (int*)malloc(N*sizeof(int));
	res = (int*)malloc(N*sizeof(int));

	//continuación reserva e inicialización
	for(i=0;i<N;i++){
		m[i] = (int*)malloc(N*sizeof(int));
		v[i] = rand()%9;
		vR[i] = 0;
		res[i] = 0;

		for(j=0;j<N;j++)
			m[i][j] = rand()%9;
	}

	
	
	if (hipInit(0) != 0){ // Obligatorio para que funcione la API
		printf("ERROR De Inicializacion\n");
		exit(0);
	}

	int idDevice = 0;
	int deviceCount;

	hipDeviceProp_t gtxProp;

	hipGetDeviceCount(&deviceCount); //Devuelve el nº de GPU's CUDA
	if (deviceCount == 0){
		printf("ERROR Ningun dispositivo es compatible con CUDA\n");
		exit(0);
	}

	//información de la GPU
	//recogemos la estrucutra de propiedades
	hipGetDeviceProperties(&gtxProp, idDevice);

	//escribimos las propiedades del dispositivo
	printf("\n\tDispositivo CUDA Instalado: \"%s\"\n\n",gtxProp.name);
	printf("\tCapacidades de Cómputo: Revision( %d.%d )\n",gtxProp.major, gtxProp.minor);

	printf("\tCantidad TOTAL de Memoria GLOBAL: %ld MBytes\n", gtxProp.totalGlobalMem/1048576);
	printf("\tCantidad de Memoria Compartida por bloque: %ld KBytes\n", gtxProp.sharedMemPerBlock/1024);
	printf("\tTamaño de WARP: %d Threads\n",gtxProp.warpSize);
	printf("\tTamaño máximo de Hilos por Bloque: %d\n",gtxProp.maxThreadsPerBlock);

	printf("\n\t\tDIMENSIONES\n\n");
	printf("\tDimensiones maximas de bloque:  ( %d x %d x %d )\n", 
		gtxProp.maxThreadsDim[0],
		gtxProp.maxThreadsDim[1],
		gtxProp.maxThreadsDim[2]);
	printf("\tDimensiones maximas de grid:   ( %d x %d x %d )\n",
	   gtxProp.maxGridSize[0],
	   gtxProp.maxGridSize[1],
	   gtxProp.maxGridSize[2]);
	printf("\tFrecuencia de Reloj: %.3f GHz\n", (float)gtxProp.clockRate/1000000);
	printf("\tMemoria CONSTANT disponible: %ld Bytes\n", gtxProp.totalConstMem);

	//reserva de momoria en DEVICE
	hipMalloc((void**)&d_m, N*sizeof(d_m));
	hipMemset(d_m, 0, N*sizeof(d_m));
	hipMalloc((void**)&d_v, N*sizeof(d_v));
	hipMemset(d_v, 0, N*sizeof(d_v));
	hipMalloc((void**)&d_vR, N*sizeof(d_vR));
	hipMemset(d_vR, 0, N*sizeof(d_vR));

	//copiamos del HOST hacia el DEVICE
	hipMemcpy(d_v, v, N*sizeof(int), hipMemcpyHostToDevice);

	//1,512
	t1GPU = omp_get_wtime();
	for(i=0;i<N;i++){
		hipMemcpy(d_m, m[i], N*sizeof(int), hipMemcpyHostToDevice);
		internProd<<<1,512>>>(d_vR, d_m, d_v);

		hipMemcpy(vR, d_vR, N*sizeof(int), hipMemcpyDeviceToHost);
		
		for(j=0;j<N;j++){
			res[i] += vR[j];
		}
	}

	t2GPU = omp_get_wtime();

	printf("\n\tRESULTADOS\n\t---------------------------------\n");
	printf("\tTiempo GPU <<<1,512>>>: %f\n", t2GPU - t1GPU);


	//2,256
	t1GPU = omp_get_wtime();
	for(i=0;i<N;i++){
		hipMemcpy(d_m, m[i], N*sizeof(int), hipMemcpyHostToDevice);
		internProd<<<2,256>>>(d_vR, d_m, d_v);

		hipMemcpy(vR, d_vR, N*sizeof(int), hipMemcpyDeviceToHost);
		
		for(j=0;j<N;j++){
			res[i] += vR[j];
		}
	}

	t2GPU = omp_get_wtime();
	printf("\tTiempo GPU <<<2,256>>>: %f\n", t2GPU - t1GPU);


	//4,128
	t1GPU = omp_get_wtime();
	for(i=0;i<N;i++){
		hipMemcpy(d_m, m[i], N*sizeof(int), hipMemcpyHostToDevice);
		internProd<<<4,128>>>(d_vR, d_m, d_v);

		hipMemcpy(vR, d_vR, N*sizeof(int), hipMemcpyDeviceToHost);
		
		for(j=0;j<N;j++){
			res[i] += vR[j];
		}
	}

	t2GPU = omp_get_wtime();
	printf("\tTiempo GPU <<<4,128>>>: %f\n", t2GPU - t1GPU);


	//8,64
	t1GPU = omp_get_wtime();
	for(i=0;i<N;i++){
		hipMemcpy(d_m, m[i], N*sizeof(int), hipMemcpyHostToDevice);
		internProd<<<8,64>>>(d_vR, d_m, d_v);

		hipMemcpy(vR, d_vR, N*sizeof(int), hipMemcpyDeviceToHost);
		
		for(j=0;j<N;j++){
			res[i] += vR[j];
		}
	}

	t2GPU = omp_get_wtime();
	printf("\tTiempo GPU <<<8,64>>>: %f\n", t2GPU - t1GPU);


	printf("\n");

	hipFree(d_m);
	hipFree(d_v);
	hipFree(d_vR);
	free(m);
	free(v);
	free(vR);
}