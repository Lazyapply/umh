#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include "colors.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N 		1000000
#define GRID	100000
#define BLOCK	100
#define M 		1000

int main(int argc, char *argv[]){

	double 	*h_x, *h_y; //variables del HOST
	double 	*d_x, *d_y; //variables del DEVICE
	int 	n 			= N;
	int 	numErr 		= 0;
	int 	DeviceCount = 0;

	//inicializacion
	printf(""YELLOW"CUDA0"RESET" ejemplo\n");

	//comprobamos que la inicializacion sea correcta
	if(hipInit(0) != hipSuccess){
		printf(""RED"ERROR de inicializacion"RESET"\n");
		exit(0);
	}
	else
		printf(""GREEN"Inicializacion correcta"RESET"\n");

	//vemos el numero de dispositivos compatibles
	hipGetDeviceCount(&DeviceCount);
	if(DeviceCount == 0){
		printf(""RED"ERROR, ningun dispositivo compatible con CUDA"RESET"\n");
		exit(0);
	}
	else
		printf(""GREEN"%d dispositivos compatibles con CUDA"RESET"\n", DeviceCount);
	
	//reservamos memoria para las matrices en HOST
	h_x = (double*)malloc(n * sizeof(h_x[0]));
	h_y = (double*)malloc(n * sizeof(h_y[0]));

	//inicializamos valores
	memset(h_x, 2, n * sizeof(h_x[0]));
	memset(h_y, 2, n * sizeof(h_y[0]));


	//reservamos para el DEVICE
	//hipMalloc(devPtr, size)
	//devPtr -> puntero para el DEVICE
	//size -> tamaño en bytes
	if(hipMalloc((void**)&d_x, n * sizeof(d_x[0])) != 0) numErr++;
	if(hipMalloc((void**)&d_y, n * sizeof(d_y[0])) != 0) numErr++;

	//comprobamos que no hay errores
	if(numErr != 0){
		printf(""RED"ERROR, reserva de memoria en DEVICE"RESET"\n");
		return EXIT_FAILURE;
	}

	//copiamos datos de memoria
	//hipMemcpy(dst, src, size_t, kind)
	//dst -> direccion de memoria de destino
	//src -> direccion de memoria origen
	//size_t -> tamaño en bytes para copiar
	//kind -> tipo de transferencia
	if(hipMemcpy(d_x, h_x, n * sizeof(d_x[0]), hipMemcpyHostToDevice) != 0)
		printf(""RED"ERROR, transferencia HOST-->DEVICE (d_x)"RESET"\n");

	if(hipMemcpy(d_y, h_x, n * sizeof(d_y[0]), hipMemcpyHostToDevice) != 0)
		printf(""RED"ERROR, transferencia HOST-->DEVICE (d_y)"RESET"\n");



	//llamada al kernel (recogemos los datos del kernel)
	if(hipMemcpy(h_x, d_x, n * sizeof(d_x[0]), hipMemcpyDeviceToHost) != 0)
		printf(""RED"ERROR, transferencia DEVICE-->HOST (d_x)"RESET"\n");

	if(hipMemcpy(h_x, d_y, n * sizeof(d_y[0]), hipMemcpyDeviceToHost) != 0)
		printf(""RED"ERROR, transferencia DEVICE-->HOST (d_y)"RESET"\n");

	//liberamos memoria del HOST
	free(h_x);
	free(h_y);

	//liberamos memoria del DEVICE
	hipFree(d_x);
	hipFree(d_y);

	return EXIT_SUCCESS;
}