#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include "colors.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N 		1000000
#define GRID	100000
#define BLOCK	100
#define M 		1000

int main(int argc, char *argv[]){

	double 	*h_x, *h_y; //variables del HOST
	double 	*d_x, *d_y; //variables del DEVICE
	int 	n 			= N;
	int 	numErr 		= 0;
	int 	DeviceCount = 0;

	//reservamos memoria para el HOST
	h_x = (double*) malloc(n * sizeof(h_x[0]));
	h_y = (double*) malloc(n * sizeof(h_y[0]));

	//inicializamos las variables
	memset(h_x, 2, n * sizeof(h_x[0]));
	memset(h_y, 2, n * sizeof(h_x[0]));

	if(hipMalloc((void**)&d_x, n * sizeof(d_x[0])) != 0)
		numErr++;
	if(hipMalloc((void**)&d_y, n * sizeof(d_y[0])) != 0)
		numErr++;

	if(numErr !=0){
		printf("ERROR reserva de memoria device %d\n", numErr);
		return EXIT_FAILURE;
	}
	else
		printf(""GREEN"Todo ok"RESET"\n");

}