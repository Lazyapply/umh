#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



//device (1)
__global__ void suma_2_enteros(int *d1, int *d2, int *sum){
	*sum = *d1 + *d2;
}

//HOST
int main(int argc, char **argv){

	int DeviceCount = 0;
	int h_d1, h_d2, h_sum; //HOST
	int *d_d1, *d_d2, *d_sum; //DEVICE (2)

	h_d1 = atoi(argv[1]);
	h_d2 = atoi(argv[2]);

	//inicializamos CUDA
	if(hipInit(0) != 0){
		printf("ERROR en la inicializacion de CUDA\n");
		exit(0);
	}

	hipGetDeviceCount(&DeviceCount);
	if(DeviceCount == 0){
		printf("ERROR, ningun dispositivo compatible con CUDA\n");
		exit(0);
	}


	//reservamos memoria en DEVICE (2.1)
	hipMalloc((void**)&d_d1, sizeof(d_d1));
	hipMalloc((void**)&d_d2, sizeof(d_d2));
	hipMalloc((void**)&d_sum, sizeof(d_sum));

	//copiamos desde HOST --> DEVICE (3)
	hipMemcpy(d_d1, &h_d1, sizeof(h_d1), hipMemcpyHostToDevice);
	hipMemcpy(d_d2, &h_d2, sizeof(h_d2), hipMemcpyHostToDevice);


	//llamamos al KERNEL
	suma_2_enteros<<<1,1,0,0>>>(d_d1, d_d2, d_sum);


	//recogemos los resultados del DEVICE (DEVICE --> HOST) (4)
	hipMemcpy(&h_sum, d_sum, sizeof(h_sum), hipMemcpyDeviceToHost);

	printf("Resultado: %d\n", h_sum);

	hipFree(d_d1);
	hipFree(d_d2);
	hipFree(d_sum);
}