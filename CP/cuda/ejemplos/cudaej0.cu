
/* Includes, system */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

/* Includes, cuda */
#include <hip/hip_runtime.h>


#define N 1000000
#define GRID 100000
#define BLOCK 100
#define M 1000

/* Main */
int main(int argc, char** argv)
{

    double *h_x,*h_y;
    double *d_x,*d_y;
    int n=N,num_err=0;
    int DeviceCount = 0;

    /* Initialize CUDA */
    printf("CUDA 0  ejemplo..\n");

    if (hipInit(0) != hipSuccess){
        printf("ERROR de inicializacion\n");
        exit(0);
	}
    hipGetDeviceCount(&DeviceCount);
    if (DeviceCount == 0){
        printf("ERROR ningun dispositivo soporta CUDA\n");
        exit(0);
	}
    /* Allocate host memory for the matrices */
    h_x = (double*)malloc(n * sizeof(h_x[0]));
    h_y = (double*)malloc(n * sizeof(h_y[0]));

    memset(h_x,2,n * sizeof(h_x[0]));
    memset(h_y,4,n * sizeof(h_x[0]));

    if (hipMalloc((void**)&d_x,n*sizeof(d_x[0])) != 0) num_err++;
    if (hipMalloc((void**)&d_y,n*sizeof(d_y[0])) != 0) num_err++;

    if (num_err != 0) {
        printf ("ERROR reserva memoria device: %d\n",num_err);
        return EXIT_FAILURE;
    }

    if (hipMemcpy(d_x,h_x,n*sizeof(d_x[0]),hipMemcpyHostToDevice) != 0) printf("ERROR1") ;
    if (hipMemcpy(d_y,h_y,n*sizeof(d_y[0]),hipMemcpyHostToDevice) != 0) printf("ERROR2") ;

    //LLAMADA A KERNEL QUE MODIFICA MEMORIA DEVICE	

    if (hipMemcpy(h_x,d_x,n*sizeof(h_x[0]),hipMemcpyDeviceToHost) != 0) printf("ERROR3") ;
    if (hipMemcpy(h_y,d_y,n*sizeof(h_y[0]),hipMemcpyDeviceToHost) != 0) printf("ERROR4") ;

    free(h_x);
    free(h_y);

    hipFree(d_x);
    hipFree(d_y);

    return EXIT_SUCCESS;
}
