
/* Includes, system */
#include <stdio.h>

#include <hip/hip_runtime.h>

#define N 1024

/* DEVICE CODE */
__global__ void suma_2_enteros(int *d1, int *d2, int *sum){

        int idBloque = blockIdx.y * gridDim.x + blockIdx.x;
        int idThread = idBloque * blockDim.z * blockDim.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;
	sum[idThread] = d1[idThread] + d2[idThread];
}

/* HOST CODE*/
int main(int argc, char** argv)
{
    int DeviceCount = 0,i;
    int *h_d1,*h_d2,*h_sum;
    int *d_d1,*d_d2,*d_sum;
    dim3 dimGrid(8,2);
    dim3 dimBlock(8,4,2);
 
	
    	

    h_d1 = (int*)malloc(N * sizeof(h_d1[0]));
    h_d2 = (int*)malloc(N * sizeof(h_d2[0]));
    h_sum = (int*)malloc(N * sizeof(h_sum[0]));	


    for (i=0;i<N;i++){h_d1[i]=i;h_d2[i]=10*i;h_sum[i]=0;}	
    /* Initialize CUDA */
    if (hipInit(0) != 0){
        printf("ERROR de inicializacion\n");
        exit(0);
        }
    hipGetDeviceCount(&DeviceCount);
    if (DeviceCount == 0){
        printf("ERROR ningun dispositivo soporta CUDA\n");
        exit(0);
        }
    hipMalloc((void**)&d_d1,N*sizeof(d_d1));hipMemset(d_d1,0,N*sizeof(d_d1));
    hipMalloc((void**)&d_d2,N*sizeof(d_d2));hipMemset(d_d2,0,N*sizeof(d_d2));
    hipMalloc((void**)&d_sum,N*sizeof(d_sum));hipMemset(d_sum,0,N*sizeof(d_sum));

    hipMemcpy(d_d1,h_d1,N*sizeof(h_d1[0]),hipMemcpyHostToDevice);
    hipMemcpy(d_d2,h_d2,N*sizeof(h_d2[0]),hipMemcpyHostToDevice);

     suma_2_enteros<<<dimGrid,dimBlock>>>(d_d1,d_d2,d_sum);

     hipMemcpy(h_sum,d_sum,N*sizeof(h_sum[0]),hipMemcpyDeviceToHost);

     for (i=510;i<520;i++) printf("Resultado: %d \n",h_sum[i]);
     hipFree(d_d1);hipFree(d_d2);hipFree(d_sum);
}