
/* Includes, system */
#include <stdio.h>

#include <hip/hip_runtime.h>

#define N 1024
#define BLOCK_SIZE 512

/* DEVICE CODE */

__global__ void IntProd(int* intprodParcial,int* d1,int* d2){

    __shared__ double accumResult[BLOCK_SIZE];

    int pos = blockIdx.x * blockDim.x + threadIdx.x;
    int iAccum = threadIdx.x;	
    int vec =  blockIdx.x;

//    accumResult[iAccum] = d1[pos] * d2[pos];
    accumResult[iAccum] = d1[pos];

    for(int stride = BLOCK_SIZE / 2; stride > 0; stride >>= 1){
        __syncthreads();
        accumResult[iAccum] += accumResult[stride + iAccum];
    }

    if(threadIdx.x == 0) {
        intprodParcial[vec] = accumResult[0];
    }
}



__global__ void suma_2_enteros(int *d1, int *d2, int *sum){

        int idBloque = blockIdx.y * gridDim.x + blockIdx.x;
        int idThread = idBloque * blockDim.z * blockDim.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;
	sum[idThread] = d1[idThread] + d2[idThread];
}

/* HOST CODE*/
int main(int argc, char** argv)
{
    int DeviceCount = 0,i;
    int *h_d1,*h_d2,*h_sum;
    int *d_d1,*d_d2,*d_sum;
 
	
    	

    h_d1 = (int*)malloc(N * sizeof(h_d1[0]));
    h_d2 = (int*)malloc(N * sizeof(h_d2[0]));
    h_sum = (int*)malloc(N * sizeof(h_sum[0]));	


    for (i=0;i<N;i++){h_d1[i]=i;h_d2[i]=i;h_sum[i]=0;}	
    /* Initialize CUDA */
    if (hipInit(0) != 0){
        printf("ERROR de inicializacion\n");
        exit(0);
        }
    hipGetDeviceCount(&DeviceCount);
    if (DeviceCount == 0){
        printf("ERROR ningun dispositivo soporta CUDA\n");
        exit(0);
        }
    hipMalloc((void**)&d_d1,N*sizeof(d_d1));hipMemset(d_d1,0,N*sizeof(d_d1));
    hipMalloc((void**)&d_d2,N*sizeof(d_d2));hipMemset(d_d2,0,N*sizeof(d_d2));
    hipMalloc((void**)&d_sum,N*sizeof(d_sum));hipMemset(d_sum,0,N*sizeof(d_sum));

    hipMemcpy(d_d1,h_d1,N*sizeof(h_d1[0]),hipMemcpyHostToDevice);
    hipMemcpy(d_d2,h_d2,N*sizeof(h_d2[0]),hipMemcpyHostToDevice);

     IntProd<<<2,BLOCK_SIZE>>>(d_sum,d_d1,d_d2);

     hipMemcpy(h_sum,d_sum,N*sizeof(h_sum[0]),hipMemcpyDeviceToHost);

     printf("Resultado: %d \n",h_sum[0]+h_sum[1]);
     hipFree(d_d1);hipFree(d_d2);hipFree(d_sum);
}
