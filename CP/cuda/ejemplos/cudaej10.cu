
/* Includes, system */
#include <stdio.h>

#include <hip/hip_runtime.h>

#define N 512

/* DEVICE CODE */
__global__ void suma_2_enteros(int *d1, int *d2, int *sum){
	sum[threadIdx.x] = d1[threadIdx.x] + d2[threadIdx.x];
}

/* HOST CODE*/
int main(int argc, char** argv)
{
    int DeviceCount = 0,i;
    int *h_d1,*h_d2,*h_sum;
    int *d_d1,*d_d2,*d_sum;

    h_d1 = (int*)malloc(N * sizeof(h_d1[0]));
    h_d2 = (int*)malloc(N * sizeof(h_d2[0]));
    h_sum = (int*)malloc(N * sizeof(h_sum[0]));

    for (i=0;i<N;i++){h_d1[i]=i;h_d2[i]=10*i;h_sum[i]=0;}	
    /* Initialize CUDA */
    if (hipInit(0) != 0){
        printf("ERROR de inicializacion\n");
        exit(0);
        }
    hipGetDeviceCount(&DeviceCount);
    if (DeviceCount == 0){
        printf("ERROR ningun dispositivo soporta CUDA\n");
        exit(0);
        }
    hipMalloc((void**)&d_d1,N*sizeof(d_d1));hipMemset(d_d1,0,N*sizeof(d_d1));
    hipMalloc((void**)&d_d2,N*sizeof(d_d2));hipMemset(d_d2,0,N*sizeof(d_d2));
    hipMalloc((void**)&d_sum,N*sizeof(d_sum));hipMemset(d_sum,0,N*sizeof(d_sum));

    hipMemcpy(d_d1,h_d1,N*sizeof(h_d1[0]),hipMemcpyHostToDevice);
    hipMemcpy(d_d2,h_d2,N*sizeof(h_d2[0]),hipMemcpyHostToDevice);

     suma_2_enteros<<<1,512>>>(d_d1,d_d2,d_sum);

     hipMemcpy(h_sum,d_sum,N*sizeof(h_sum[0]),hipMemcpyDeviceToHost);

     for (i=0;i<10;i++) printf("Resultado: %d \n",h_sum[i]);
     hipFree(d_d1);hipFree(d_d2);hipFree(d_sum);
}
