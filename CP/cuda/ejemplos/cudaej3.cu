
/* Includes, system */
#include <stdio.h>

#include <hip/hip_runtime.h>


/* DEVICE CODE */
__global__ void primer_kernel(){
}

/* HOST CODE*/
int main(int argc, char** argv)
{
    int DeviceCount = 0;

    /* Initialize CUDA */
    if (hipInit(0) != 0){
        printf("ERROR de inicializacion\n");
        exit(0);
        }
    hipGetDeviceCount(&DeviceCount);
    if (DeviceCount == 0){
        printf("ERROR ningun dispositivo soporta CUDA\n");
        exit(0);
        }
  primer_kernel<<<1,1,0,0>>>();
  printf("Se dispone de %d unidade(s) GPU.\n",DeviceCount);

}
