
/* Includes, system */
#include <stdio.h>

#include <hip/hip_runtime.h>


/* DEVICE CODE */
__global__ void suma_2_enteros(int *d1, int *d2, int *sum){
	*sum = *d1 + *d2;
}

/* HOST CODE*/
int main(int argc, char** argv)
{
    int DeviceCount = 0;
    int h_d1,h_d2,h_sum;
    int *d_d1,*d_d2,*d_sum;

    h_d1 = 2; h_d2 = 3;
    /* Initialize CUDA */
    if (hipInit(0) != 0){
        printf("ERROR de inicializacion\n");
        exit(0);
        }
    hipGetDeviceCount(&DeviceCount);
    if (DeviceCount == 0){
        printf("ERROR ningun dispositivo soporta CUDA\n");
        exit(0);
        }

    hipMalloc((void**)&d_d1, sizeof(d_d1));
    hipMalloc((void**)&d_d2, sizeof(d_d2));
    hipMalloc((void**)&d_sum, sizeof(d_sum));


    hipMemcpy(d_d1,&h_d1,sizeof(h_d1),hipMemcpyHostToDevice);
    hipMemcpy(d_d2,&h_d2,sizeof(h_d2),hipMemcpyHostToDevice);

     suma_2_enteros<<<1,1>>>(d_d1,d_d2,d_sum);

     hipMemcpy(&h_sum,d_sum,sizeof(h_sum),hipMemcpyDeviceToHost);

     printf("Resultado: %d \n",h_sum);
     hipFree(d_d1);hipFree(d_d2);hipFree(d_sum);
}
