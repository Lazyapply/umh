
/* Includes, system */
#include <stdio.h>

#include <hip/hip_runtime.h>


/* DEVICE CODE */
__global__ void suma_2_enteros(int *d1, int *d2, int *sum){
	*sum = *d1 + *d2;
}

/* HOST CODE*/
int main(int argc, char** argv)
{
    int DeviceCount = 0;
    int h_d1,h_d2,h_sum;
    h_d1 = 2; h_d2 = 3;


    /* Initialize CUDA */
    if (hipInit(0) != 0){
        printf("ERROR de inicializacion\n");
        exit(0);
        }
    hipGetDeviceCount(&DeviceCount);
    if (DeviceCount == 0){
        printf("ERROR ningun dispositivo soporta CUDA\n");
        exit(0);
        }
  suma_2_enteros<<<1,1>>>(&h_d1,&h_d2,&h_sum);
  printf("Resultado: %d \n",h_sum);

}
