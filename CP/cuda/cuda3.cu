#include <stdio.h>
#include <hip/hip_runtime.h>


//device
__global__ void primer_kernel(){}


//host
int main(int argc, char *argv[]){
	int DeviceCount = 0;

	//inicializamos CUDA
	if(hipInit(0) != 0){
		printf("ERROR en la inicializacion\n");
		exit(0);
	}

	//obtenemos el numero de dispositivos compatibles con CUDA
	hipGetDeviceCount(&DeviceCount);
	if(DeviceCount == 0){
		printf("ERROR, ningun dispositivo soporta CUDA\n");
		return EXIT_FAILURE;
	}

	//llamamos al codigo del kernel
	primer_kernel<<<1,1,0,0>>>();
	printf("Se dispone de %d unidad(es) GPU\n", DeviceCount);
}